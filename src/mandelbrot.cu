
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>


__device__ hipDoubleComplex cpow(hipDoubleComplex z, int power) {
    hipDoubleComplex result = make_hipDoubleComplex(1.0, 0.0);
    for (int i = 0; i < power; i++) {
        result = hipCmul(result, z);
    }
    return result;
}

__global__ void iterate_mandelbrot(hipDoubleComplex* c, int* iterations, int width, int height, int max_iterations, int power)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= width || idy >= height) return;
    int index = idy * width + idx;

    hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
    int iter;

    for (iter = 0; iter < max_iterations; iter++) {
        if (hipCabs(z) > 2.0) break;
        
        z = hipCadd(cpow(z, power), c[index]);
    }

    iterations[index] = iter;
}


