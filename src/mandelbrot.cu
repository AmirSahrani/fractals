#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <mandelbrot.c>
#include <hip/hip_vector_types.h>

__device__ hipDoubleComplex cpow(hipDoubleComplex z, int power) {
  hipDoubleComplex result = make_hipDoubleComplex(1.0, 0.0);
  for (int i = 0; i < power; i++) {
    result = hipCmul(result, z);
  }
  return result;
}

__device__ int iterate_mandelbrotGPU(hipDoubleComplex start_c) {
  float bound = 2;
  hipDoubleComplex z = start_c;
  int iter;

  for (iter = 0; (iter < MAX_ITERATIONS); iter++) {
    z = hipCadd(cpow((hipDoubleComplex)z, 5.0), start_c);
    if (hipCabs(z) > 2.0) {
      return iter;
    }
  };
  return MAX_ITERATIONS;
}

__global__ void GPUIterations(double complex_plane[SCREENWIDTH][2],
                              int *results, int width, int height,
                              int max_iterations) {
  int index = blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < width * height; i += stride) {
    if (i < SCREENWIDTH * 2) {
      hipDoubleComplex c =
          make_hipDoubleComplex(complex_plane[i][0], complex_plane[i][1]);
      results[i] = iterate_mandelbrotGPU(c);
    }
  }
}
