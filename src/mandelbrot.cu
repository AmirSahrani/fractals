#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <hip/hip_vector_types.h>
#include "mandelbrot.h"

__device__ hipDoubleComplex cupow(hipDoubleComplex z, int power) {
  hipDoubleComplex result = make_hipDoubleComplex(1.0, 0.0);
  for (int i = 0; i < power; i++) {
    result = hipCmul(result, z);
  }
  return result;
}

__device__ int iterate_mandelbrotGPU(hipDoubleComplex start_c) {
  float bound = 2;
  hipDoubleComplex z = start_c;
  int iter;

  for (iter = 0; (iter < MAX_ITERATIONS); iter++) {
    z = hipCadd(cupow((hipDoubleComplex)z, 5.0), start_c);
    if (hipCabs(z) > exponent) {
      return iter;
    }
  };
  return MAX_ITERATIONS;
}

__global__ void GPUIterations(double complex_plane[SCREENWIDTH][2],
                              int *results, int width, int height,
                              int max_iterations) {
  int index = blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < width * height; i += stride) {
    if (i < SCREENWIDTH * 2) {
      hipDoubleComplex c =
          make_hipDoubleComplex(complex_plane[i][0], complex_plane[i][1]);
      results[i] = iterate_mandelbrotGPU(c);
    }
  }
}
